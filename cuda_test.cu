// main.cu
/* Compile and run with:
nvcc main.cu -o run
./run
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__ void add(int n, float* x, float* y) {
    // At each index, add x to y.
    for (int i = 0; i < n; i++) {
        y[i] = x[i] + y[i];
    }
}

int main(void) {
    int N = 100;
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // Initialize our x and y arrays with some floats.
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run the function on using the GPU.
    // <<NumBlocks, NumThreadsPerBlock>>
    add<<<1, 1>>>(N, x, y); // Notice the brackets.

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}